#include "hip/hip_runtime.h"
/*
Copyright (c) 2014, Texas State University. All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted for academic, research, experimental, or personal use provided
that the following conditions are met:

   * Redistributions of source code must retain the above copyright notice,
     this list of conditions and the following disclaimer.
   * Redistributions in binary form must reproduce the above copyright notice,
     this list of conditions and the following disclaimer in the documentation
     and/or other materials provided with the distribution.
   * Neither the name of Texas State University nor the names of its
     contributors may be used to endorse or promote products derived from this
     software without specific prior written permission.

For all other uses, please contact the Office for Commercialization and Industry
Relations at Texas State University <http://www.txstate.edu/ocir/>.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/


#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// no point in using precise FP math or double precision as we are rounding
// the results to the nearest integer anyhow

/******************************************************************************/
/*** 2-opt with random restarts ***********************************************/
/******************************************************************************/

#define tilesize 128
#define dist(a, b) __float2int_rn(sqrtf((px[a] - px[b]) * (px[a] - px[b]) + (py[a] - py[b]) * (py[a] - py[b])))
#define swap(a, b) {float tmp = a;  a = b;  b = tmp;}

static __device__ int climbs_d;
static __device__ volatile int best_d;
static __device__ int lock_d;
static __device__ float *soln_d;
extern __shared__ int buf_s[];

static __global__ void Init()
{
  climbs_d = 0;
  best_d = INT_MAX;
  lock_d = 0;
  soln_d = NULL;
}

static __global__ //__launch_bounds__(1024, 2)
void TwoOpt(int cities, float *posx_d, float *posy_d, int *glob_d)
{
//each block handles a a hill climb.


  //access a particular restart / initial tour from global memory
  int *buf = &glob_d[blockIdx.x * ((3 * cities + 2 + 31) / 32 * 32)];
  float *px = (float *)(&buf[cities]);
  float *py = &px[cities + 1];
  __shared__ float px_s[tilesize];
  __shared__ float py_s[tilesize];
  __shared__ int bf_s[tilesize];
//copy posx_d/posy_d to px/py. glob_d was not initialize and holds all restarts.
  for (int i = threadIdx.x; i < cities; i += blockDim.x) px[i] = posx_d[i];
  for (int i = threadIdx.x; i < cities; i += blockDim.x) py[i] = posy_d[i];
  __syncthreads();


//randomize the current block's path
  if (threadIdx.x == 0) {  // serial permutation
    hiprandState rndstate;
    hiprand_init(blockIdx.x, 0, 0, &rndstate);
    for (int i = 1; i < cities; i++) {
      int j = hiprand(&rndstate) % (cities - 1) + 1;
      swap(px[i], px[j]);
      swap(py[i], py[j]);
    }
    px[cities] = px[0];
    py[cities] = py[0];
  }
  __syncthreads();




//the biggest and main while loop. performs the whole hill climb, until a local optimum is reached (minchange>=0).
  int minchange;
  do {
    //fill buf[] with distances.
    for (int i = threadIdx.x; i < cities; i += blockDim.x) buf[i] = -dist(i, i + 1);
    __syncthreads();

    minchange = 0;
    int mini = 1;
    int minj = 0;
    for (int ii = 0; ii < cities - 2; ii += blockDim.x) {
      int i = ii + threadIdx.x;
      float pxi0, pyi0, pxi1, pyi1, pxj1, pyj1;
      if (i < cities - 2) {
        minchange -= buf[i];
        pxi0 = px[i];
        pyi0 = py[i];
        pxi1 = px[i + 1];
        pyi1 = py[i + 1];
        pxj1 = px[cities];
        pyj1 = py[cities];
      }
      for (int jj = cities - 1; jj >= ii + 2; jj -= tilesize) {
        int bound = jj - tilesize + 1;
        //copy to shared memory
	for (int k = threadIdx.x; k < tilesize; k += blockDim.x) {
          if (k + bound >= ii + 2) {
            px_s[k] = px[k + bound];
            py_s[k] = py[k + bound];
            bf_s[k] = buf[k + bound];
          }
        }
        __syncthreads();

        int lower = bound;
        if (lower < i + 2) lower = i + 2;
        for (int j = jj; j >= lower; j--) {
          int jm = j - bound;
          float pxj0 = px_s[jm];
          float pyj0 = py_s[jm];
          int change = bf_s[jm]
            + __float2int_rn(sqrtf((pxi0 - pxj0) * (pxi0 - pxj0) + (pyi0 - pyj0) * (pyi0 - pyj0)))
            + __float2int_rn(sqrtf((pxi1 - pxj1) * (pxi1 - pxj1) + (pyi1 - pyj1) * (pyi1 - pyj1)));
          pxj1 = pxj0;
          pyj1 = pyj0;
          if (minchange > change) {
            minchange = change;
            mini = i;
            minj = j;
          }
        }
        __syncthreads();
      }

      if (i < cities - 2) {
        minchange += buf[i];
      }
    }
    __syncthreads();

    int change = buf_s[threadIdx.x] = minchange;
    if (threadIdx.x == 0) atomicAdd(&climbs_d, 1);  // stats only
    __syncthreads();

    int j = blockDim.x;
    do {
      int k = (j + 1) / 2;
      if ((threadIdx.x + k) < j) {
        int tmp = buf_s[threadIdx.x + k];
        if (change > tmp) change = tmp;
        buf_s[threadIdx.x] = change;
      }
      j = k;
      __syncthreads();
    } while (j > 1);

    if (minchange == buf_s[0]) {
      buf_s[1] = threadIdx.x;  // non-deterministic winner
    }
    __syncthreads();

    if (threadIdx.x == buf_s[1]) {
      buf_s[2] = mini + 1;
      buf_s[3] = minj;
    }
    __syncthreads();

    minchange = buf_s[0];
    mini = buf_s[2];
    int sum = buf_s[3] + mini;
    for (int i = threadIdx.x; (i + i) < sum; i += blockDim.x) {
      if (mini <= i) {
        int j = sum - i;
        swap(px[i], px[j]);
        swap(py[i], py[j]);
      }
    }
    __syncthreads();
  } while (minchange < 0);




//add up the distances 'belonging' to a thread (preparing for reduction to find tour length)
  int term = 0;
  for (int i = threadIdx.x; i < cities; i += blockDim.x) {
    term += dist(i, i + 1);
  }
  buf_s[threadIdx.x] = term;
  __syncthreads();

//a reduction to find tour length.  
int j = blockDim.x;
  do {
    int k = (j + 1) / 2;
    if ((threadIdx.x + k) < j) {
      term += buf_s[threadIdx.x + k];
    }
    __syncthreads();
    if ((threadIdx.x + k) < j) {
      buf_s[threadIdx.x] = term;
    }
    j = k;
    __syncthreads();
  } while (j > 1);
//check if computed tour is best. get soln_d and best_d.
  if (threadIdx.x == 0) {
    atomicMin((int *)&best_d, term);
    if (best_d == term) {
      while (atomicExch(&lock_d, 1) != 0);  // acquire
      if (best_d == term) {
        soln_d = px;
      }
      lock_d = 0;  // release
      __threadfence();
    }
  }
}

/******************************************************************************/
/*** find best thread count ***************************************************/
/******************************************************************************/

static int best_thread_count_kepler(int cities)
{
  int max, best, threads, smem, blocks, thr, perf, bthr;

  max = cities - 2;
  if (max > 1024) max = 1024;
  best = 0;
  bthr = 4;
  for (threads = 1; threads <= max; threads++) {
    smem = sizeof(int) * threads + 2 * sizeof(float) * tilesize + sizeof(int) * tilesize;
    blocks = (16384 * 2) / smem;
    if (blocks > 16) blocks = 16;
    thr = (threads + 31) / 32 * 32;
    while (blocks * thr > 2048) blocks--;
    //while (blocks * thr > 1536) blocks--;
    perf = threads * blocks;
    if (perf > best) {
      best = perf;
      bthr = threads;
    }
  }

  return bthr;
}

static int best_thread_count(int cities)
{
  int max, best, threads, smem, blocks, thr, perf, bthr;

  max = cities - 2;
  if (max > 1024) max = 1024;
  best = 0;
  bthr = 4;
  for (threads = 1; threads <= max; threads++) {
    smem = sizeof(int) * threads + 2 * sizeof(float) * tilesize + sizeof(int) * tilesize;
    blocks = (16384 * 2) / smem;
    if (blocks > 8) blocks = 8;
    thr = (threads + 31) / 32 * 32;
    //while (blocks * thr > 2048) blocks--;
    while (blocks * thr > 1536) blocks--;
    perf = threads * blocks;
    if (perf > best) {
      best = perf;
      bthr = threads;
    }
  }

  return bthr;
}


/******************************************************************************/
/*** helper code **************************************************************/
/******************************************************************************/

static void CudaTest(char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}

#define mallocOnGPU(addr, size) if (hipSuccess != hipMalloc((void **)&addr, size)) fprintf(stderr, "could not allocate GPU memory\n");  CudaTest("couldn't allocate GPU memory");
#define copyToGPU(to, from, size) if (hipSuccess != hipMemcpy(to, from, size, hipMemcpyHostToDevice)) fprintf(stderr, "copying of data to device failed\n");  CudaTest("data copy to device failed");
#define copyFromGPU(to, from, size) if (hipSuccess != hipMemcpy(to, from, size, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of data from device failed\n");  CudaTest("data copy from device failed");
#define copyFromGPUSymbol(to, from, size) if (hipSuccess != hipMemcpyFromSymbol(to, HIP_SYMBOL(from), size)) fprintf(stderr, "copying of symbol from device failed\n");  CudaTest("symbol copy from device failed");
#define copyToGPUSymbol(to, from, size) if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(to), from, size)) fprintf(stderr, "copying of symbol to device failed\n");  CudaTest("symbol copy to device failed");

/******************************************************************************/
/*** read TSPLIB input ********************************************************/
/******************************************************************************/

static int readInput(char *fname, float **posx_d, float **posy_d)  // ATT and CEIL_2D edge weight types are not supported
{
  int ch, cnt, in1, cities;
  float in2, in3;
  FILE *f;
  float *posx, *posy;
  char str[256];  // potential for buffer overrun

  f = fopen(fname, "rt");
  if (f == NULL) {fprintf(stderr, "could not open file %s\n", fname);  exit(-1);}

  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

  ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
  fscanf(f, "%s\n", str);
  cities = atoi(str);
fprintf(stderr,"Cities read: %d\n",cities);
  if (cities <= 2) {fprintf(stderr, "only %d cities\n", cities);  exit(-1);}

  posx = (float *)malloc(sizeof(float) * cities);  if (posx == NULL) {fprintf(stderr, "cannot allocate posx\n");  exit(-1);}
  posy = (float *)malloc(sizeof(float) * cities);  if (posy == NULL) {fprintf(stderr, "cannot allocate posy\n");  exit(-1);}

  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  fscanf(f, "%s\n", str);
  if (strcmp(str, "NODE_COORD_SECTION") != 0) {fprintf(stderr, "wrong file format\n");  exit(-1);}

  cnt = 0;
  while (fscanf(f, "%d %f %f\n", &in1, &in2, &in3)) {  
  //while (cnt <cities){
	if(cnt >=cities) break;  
  //if(cnt == 0) { fprintf(stderr,"%d %f %f\n",in1,in2,in3); }
	posx[cnt] = in2;
    posy[cnt] = in3;
++cnt;    
if (cnt > cities) {fprintf(stderr, "input too long (%d, %d %f %f)\n",cnt,in1,in2,in3);  exit(-1);}
    if (cnt != in1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, in1);  exit(-1);}
	//++cnt;  
}
  if (cnt != cities) {fprintf(stderr, "read %d instead of %d cities\n", cnt, cities);  exit(-1);}

  //fscanf(f, "%s", str);
  //if (strcmp(str, "EOF") != 0) {fprintf(stderr, "didn't see 'EOF' at end of file\n");  exit(-1);}

  mallocOnGPU(*posx_d, sizeof(float) * cities);
  mallocOnGPU(*posy_d, sizeof(float) * cities);
  copyToGPU(*posx_d, posx, sizeof(float) * cities);
  copyToGPU(*posy_d, posy, sizeof(float) * cities);

  fclose(f);
  free(posx);
  free(posy);

  return cities;
}

/******************************************************************************/
/*** main function ************************************************************/
/******************************************************************************/

int main(int argc, char *argv[])
{
  printf("2-opt TSP CUDA GPU code v2.2 [Kepler]\n");
  printf("Copyright (c) 2014, Texas State University. All rights reserved.\n");

  int cities, restarts, climbs, best, threads;
  long long moves;
  int *glob_d;
  float *posx_d, *posy_d, *soln;
  double runtime;
  struct timeval starttime, endtime;

  if (argc != 3) {fprintf(stderr, "\narguments: input_file restart_count\n"); exit(-1);}
  cities = readInput(argv[1], &posx_d, &posy_d);
  restarts = atoi(argv[2]);
  if (restarts < 1) {fprintf(stderr, "restart_count is too small: %d\n", restarts); exit(-1);}

  printf("configuration: %d cities, %d restarts, %s input\n", cities, restarts, argv[1]);

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(TwoOpt), hipFuncCachePreferEqual);

  if (100 > cities) {
    fprintf(stderr, "the problem size is too small for this version of the code\n");
  } else {
    threads = best_thread_count_kepler(cities);
    mallocOnGPU(glob_d, 4 * restarts * ((3 * cities + 2 + 31) / 32 * 32));

    gettimeofday(&starttime, NULL);
    Init<<<1, 1>>>();
	printf("Thread count: %d\n",threads);	
    TwoOpt<<<restarts, threads, sizeof(int) * threads>>>(cities, posx_d, posy_d, glob_d);
    CudaTest("kernel launch failed");  // needed for timing
hipDeviceSynchronize();    
gettimeofday(&endtime, NULL);

    copyFromGPUSymbol(&climbs, climbs_d, sizeof(int));
    copyFromGPUSymbol(&best, best_d, sizeof(int));
    copyFromGPUSymbol(&soln, soln_d, sizeof(void *));
    float *pos = (float *)malloc(sizeof(float) * (cities + 1) * 2);  if (pos == NULL) {fprintf(stderr, "cannot allocate pos\n");  exit(-1);}
    copyFromGPU(pos, soln, sizeof(float) * (cities + 1) * 2);

    runtime = endtime.tv_sec + endtime.tv_usec / 1000000.0 - starttime.tv_sec - starttime.tv_usec / 1000000.0;
    moves = 1LL * climbs * (cities - 2) * (cities - 1) / 2;

    fprintf(stderr,"runtime = %.4f s, %.3f Gmoves/s\n", runtime, moves * 0.000000001 / runtime);
    fprintf(stderr,"best found tour length = %d\n", best);
    if (1) {  // print best found solution
      for (int i = 0; i < cities; i++) {
        //printf("%.1f %.1f\n", pos[i], pos[i + cities + 1]);
      }
    }

    fflush(stdout);
    hipFree(glob_d);
  }

  hipFree(posx_d);
  hipFree(posy_d);
  return 0;
}
