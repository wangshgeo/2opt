#include "hip/hip_runtime.h"
#include "include/instance.h"
#include "driver.h"
#include "cuda_utils.h"
#include "opt2gpu.h"

#include "parameters.h"

#include "distanceFunctions.h"

dtype reduceCpu(dtype*h_A,unsigned int N);

__device__ inline int kfromij(int i,int j)
{
	i-=1;
	j-=2;
	return i+((j*(j+1))>>1);
}
inline void ijfromk(int k,int *ij)
{
	int i = (int)(((-1+sqrtf(1+4*2*k)))/2);//floating point calculation!
	int j = k-((i*(i+1))>>1);
	ij[0] = j+1;
	ij[1] = i+2;
}
__device__ inline void d_ijfromk(int k,int *ij)
{
	int i = (int)(((-1+sqrtf(1+4*2*k)))/2);//floating point calculation!
	int j = k-((i*(i+1))>>1);
	ij[0] = j+1;
	ij[1] = i+2;
}

__device__ void warpMinReduce2(volatile dtype*sdata,volatile int*smink,int tid)
{
	dtype currentsdata = sdata[tid];
	int currentsmink = smink[tid];
	dtype nextsdata = sdata[tid+32];
	int nextsmink = smink[tid+32];
	int check = nextsdata<currentsdata;
	smink[tid] = currentsmink = (check)?nextsmink:currentsmink;
	sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
	nextsdata = sdata[tid+16];
	nextsmink = smink[tid+16];
	check = nextsdata<currentsdata;
	smink[tid] = currentsmink = (check)?nextsmink:currentsmink;
	sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
	nextsdata = sdata[tid+8];
	nextsmink = smink[tid+8];
	check = nextsdata<currentsdata;
	smink[tid] = currentsmink = (check)?nextsmink:currentsmink;
	sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
	nextsdata = sdata[tid+4];
	nextsmink = smink[tid+4];
	check = nextsdata<currentsdata;
	smink[tid] = currentsmink = (check)?nextsmink:currentsmink;
	sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
	nextsdata = sdata[tid+2];
	nextsmink = smink[tid+2];
	check = nextsdata<currentsdata;
	smink[tid] = currentsmink = (check)?nextsmink:currentsmink;
	sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
	nextsdata = sdata[tid+1];
	nextsmink = smink[tid+1];
	check = nextsdata<currentsdata;
	smink[tid] = currentsmink = (check)?nextsmink:currentsmink;
	sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
}

__global__ void 
kernel9 (COORD *d_coords, dtype *d_differences,int *d_mink, 
	unsigned int nc,unsigned int BB,int globalStart)
{//Version 9. Save distances for adjacent points in i and j AND between points in i and j.
	//also get the index.
	COORD ii[NCOORDS];//registers to hold coordinates
	COORD jj[NCOORDS];
	dtype savedi[TSD];//saved distances corresponding to i (i-1 to i)
	dtype savedj[TSD];//saved distances corresponding to j (j to j+1)
	dtype savedij[TSD+1][TSD+1];//saved distances (i-1 -> j)

	//Calculate the indices needed for all later calculations.
	unsigned int maxn = nc-3;
	//compute bi,bj from blockIdx.x
	int block = globalStart+blockIdx.x;
	int bi = (int)(((-1+sqrtf(1+4*2*block)))/2);//floating point calculation!
	int bj = block-((bi*(bi+1))>>1);
	bi=BB-bi-1;
	bj=BB-bj-1-bi;
	unsigned int istart = bi*BSD+1;//i start from block
	istart+=TSD*threadIdx.x;//i start from thread
	int jstart =  (nc-2)-(bj+1)*BSD+1;//j start from block
	jstart+=TSD*threadIdx.y;//j start from thread

	//Now transfer the memory from global into registers.
	#pragma unroll
	for(int c=0;c<NCOORDS;++c)
	{
		int id=istart-1+c;
		if(id<maxn+1)
		{
			ii[c] = d_coords[id];
		}
	}
	#pragma unroll
	for(int i=0;i<TSD;++i)
	{
		savedi[i]=distanceEUC_2D(ii[i].x,ii[i].y,ii[i+1].x,ii[i+1].y);
	}
	#pragma unroll
	for(int c=0;c<NCOORDS;++c)
	{
		int id=jstart+c;
		if(id<nc-bj*BSD && id>0)
		{
			jj[c] = d_coords[id];
		}
	}
	#pragma unroll
	for(int j=0;j<TSD;++j)
	{
		savedj[j]=distanceEUC_2D(jj[j].x,jj[j].y,jj[j+1].x,jj[j+1].y);
	}
	#pragma unroll
	for(int sx=0;sx<TSD+1;++sx)
	{
		#pragma unroll
		for(int sy=0;sy<TSD+1;++sy)
		{
			savedij[sx][sy]=distanceEUC_2D(ii[sx].x,ii[sx].y,jj[sy].x,jj[sy].y);
		}
	}
	//__syncthreads();

	//Now let us calculate the differences!
	dtype min = 0;
	int mink = -1;
	#pragma unroll
	for(int sx=0;sx<TSD;++sx)
	{
		#pragma unroll
		for(int sy=0;sy<TSD;++sy)
		{
			//first calculate the global indices to see if it is in calculation domain.
			int gi = istart+sx;
			int gj = jstart+sy;
			if(gi<maxn+1 && gj>gi && gj < nc-1-bj*BSD)
			{
				dtype dold = savedi[sx]+savedj[sy];
				dtype dnew = savedij[sx][sy]+savedij[sx+1][sy+1];
				//dtype dnew = distanceEUC_2D(ii[sx].x,ii[sx].y,jj[sy].x,jj[sy].y)+
				//		distanceEUC_2D(ii[sx+1].x,ii[sx+1].y,jj[sy+1].x,jj[sy+1].y);
				dtype difference = dnew-dold;
				mink = (difference<min)?(kfromij(gi,gj)):mink;
				min = (difference<min)?difference:min;
			}
		}
	}
	//__syncthreads();

	//Reduction step
	__shared__ dtype sdata[TPB2*TPB2];
	__shared__ int smink[TPB2*TPB2];
	int tid = threadIdx.x+threadIdx.y*TPB2;
	sdata[tid] = min;
	smink[tid] = mink;
	__syncthreads();
	int check=-1;
	dtype currentsdata = sdata[tid];
	dtype nextsdata;
	int currentsmink = smink[tid];
	int nextsmink;
	if (TPB2*TPB2 >= 1024){ if (tid < 512) {
		//check = sdata[tid+512]<sdata[tid];
		nextsdata = sdata[tid+512];
		nextsmink = smink[tid+512];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (TPB2*TPB2 >= 512) { if (tid < 256) {
		nextsdata = sdata[tid+256];
		nextsmink = smink[tid+256];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (TPB2*TPB2 >= 256) { if (tid < 128) {
		nextsdata = sdata[tid+128];
		nextsmink = smink[tid+128];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (TPB2*TPB2 >= 128) { if (tid < 64) {
		nextsdata = sdata[tid+64];
		nextsmink = smink[tid+64];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (tid < 32) warpMinReduce2(sdata,smink,tid);
	if (tid == 0){
		d_differences[block] = sdata[0];
		d_mink[block] = smink[0];
	}
}

__global__ void 
kernel10 (COORD *d_coords, dtype *d_differences,int *d_mink, 
	unsigned int nc,unsigned int BB,int globalStart)
{//Version 10. Save distances for adjacent points in i and j AND between points in i and j.
	//also get the index.
	//CURRENTLY BROKEN!
	COORD ii[NCOORDS];//registers to hold coordinates
	COORD jj[NCOORDS];
	dtype savedi[TSD];//saved distances corresponding to i (i-1 to i)
	dtype savedj[TSD];//saved distances corresponding to j (j to j+1)
	dtype savedij[TSD+1][TSD+1];//saved distances (i-1 -> j)

	//Calculate the indices needed for all later calculations.
	int maxn = nc-3;
	//compute bi,bj from blockIdx.x
	int block = globalStart+blockIdx.x;
	int bi = (int)(((-1+sqrtf(1+4*2*block)))/2);//floating point calculation!
	int bj = block-((bi*(bi+1))>>1);
	bi=BB-bi-1;
	bj=BB-bj-1-bi;
	int istart0 = bi*BSD+1;//i start from block
	int istart = istart0 + TSD*threadIdx.x;//i start from thread
	int jstart0 =  (nc-2)-(bj+1)*BSD+1;//j start from block
	int jstart = jstart0 + TSD*threadIdx.y;//j start from thread

	//Now transfer the memory from global into registers.
	__shared__ COORD sharedi[TSD*TPB2*TPB2];
	#pragma unroll
	for(int t=0;t<TSD;++t)
	{
		int li = t*blockDim.x + threadIdx.x;
		int id = istart0 - 1 + li;
		if(id < maxn+1)
		{
			sharedi[li] = d_coords[id];
		}
	}
	#pragma unroll
	for(int i=0;i<NCOORDS;++i)
	{
		int id = TSD*threadIdx.x + i;
		if(id < TSD*blockDim.x)
			ii[i] = sharedi[id];
	}
	/*
	//old
	#pragma unroll
	for(int c=0;c<NCOORDS;++c)
	{
		int id=istart-1+c;
		if(id<maxn+1)
		{
			ii[c] = d_coords[id];
		}
	}
	//end
	*/
	#pragma unroll
	for(int i=0;i<TSD;++i)
	{
		savedi[i]=distanceEUC_2D(ii[i].x,ii[i].y,ii[i+1].x,ii[i+1].y);
	}
	#pragma unroll
	for(int c=0;c<NCOORDS;++c)
	{
		int id=jstart+c;
		if(id<nc-bj*BSD && id>0)
		{
			jj[c] = d_coords[id];
		}
	}
	#pragma unroll
	for(int j=0;j<TSD;++j)
	{
		savedj[j]=distanceEUC_2D(jj[j].x,jj[j].y,jj[j+1].x,jj[j+1].y);
	}
	#pragma unroll
	for(int sx=0;sx<TSD+1;++sx)
	{
		#pragma unroll
		for(int sy=0;sy<TSD+1;++sy)
		{
			savedij[sx][sy]=distanceEUC_2D(ii[sx].x,ii[sx].y,jj[sy].x,jj[sy].y);
		}
	}
	//__syncthreads();

	//Now let us calculate the differences!
	dtype min = 0;
	int mink = -1;
	#pragma unroll
	for(int sx=0;sx<TSD;++sx)
	{
		#pragma unroll
		for(int sy=0;sy<TSD;++sy)
		{
			//first calculate the global indices to see if it is in calculation domain.
			int gi = istart+sx;
			int gj = jstart+sy;
			if(gi<maxn+1 && gj>gi && gj < nc-1-bj*BSD)
			{
				dtype dold = savedi[sx]+savedj[sy];
				dtype dnew = savedij[sx][sy]+savedij[sx+1][sy+1];
				//dtype dnew = distanceEUC_2D(ii[sx].x,ii[sx].y,jj[sy].x,jj[sy].y)+
				//		distanceEUC_2D(ii[sx+1].x,ii[sx+1].y,jj[sy+1].x,jj[sy+1].y);
				dtype difference = dnew-dold;
				mink = (difference<min)?(kfromij(gi,gj)):mink;
				min = (difference<min)?difference:min;
			}
		}
	}
	//__syncthreads();

	//Reduction step
	__shared__ dtype sdata[TPB2*TPB2];
	__shared__ int smink[TPB2*TPB2];
	int tid = threadIdx.x+threadIdx.y*TPB2;
	sdata[tid] = min;
	smink[tid] = mink;
	__syncthreads();
	int check=-1;
	dtype currentsdata = sdata[tid];
	dtype nextsdata;
	int currentsmink = smink[tid];
	int nextsmink;
	if (TPB2*TPB2 >= 1024){ if (tid < 512) {
		//check = sdata[tid+512]<sdata[tid];
		nextsdata = sdata[tid+512];
		nextsmink = smink[tid+512];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (TPB2*TPB2 >= 512) { if (tid < 256) {
		nextsdata = sdata[tid+256];
		nextsmink = smink[tid+256];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (TPB2*TPB2 >= 256) { if (tid < 128) {
		nextsdata = sdata[tid+128];
		nextsmink = smink[tid+128];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (TPB2*TPB2 >= 128) { if (tid < 64) {
		nextsdata = sdata[tid+64];
		nextsmink = smink[tid+64];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (tid < 32) warpMinReduce2(sdata,smink,tid);
	if (tid == 0){
		d_differences[block] = sdata[0];
		d_mink[block] = smink[0];
	}
}

__global__ void 
kernel11 (COORD *d_coords, dtype *d_differences,int *d_mink, 
	unsigned int nc,unsigned int BB,int globalStart)
{//Version 11; Version 9 but with corrections to alleviate the NVPROF-diagnosed global memory problems. 
	//  Save distances for adjacent points in i and j AND between points in i and j.
	//also get the index.
	COORD ii[NCOORDS];//registers to hold coordinates
	COORD jj[NCOORDS];
	dtype savedi[TSD];//saved distances corresponding to i (i-1 to i)
	dtype savedj[TSD];//saved distances corresponding to j (j to j+1)
	dtype savedij[TSD+1][TSD+1];//saved distances (i-1 -> j)

	//Calculate the indices needed for all later calculations.
	unsigned int maxn = nc-3;
	//compute bi,bj from blockIdx.x
	int block = globalStart+blockIdx.x;
	int bi = (int)(((-1+sqrtf(1+4*2*block)))/2);//floating point calculation!
	int bj = block-((bi*(bi+1))>>1);
	bi=BB-bi-1;
	bj=BB-bj-1-bi;
	unsigned int istart = bi*BSD+1;//i start from block
	istart+=TSD*threadIdx.x;//i start from thread
	int jstart =  (nc-2)-(bj+1)*BSD+1;//j start from block
	jstart+=TSD*threadIdx.y;//j start from thread

	//Now transfer the memory from global into registers.
	#pragma unroll
	for(int c=0;c<NCOORDS;++c)
	{
		int idi=istart-1+c;
		if(idi<maxn+1)
		{
			ii[c] = d_coords[idi];
		}
		int idj=jstart+c;
		if(idj<nc-bj*BSD && idj>0)
		{
			jj[c] = d_coords[idj];
		}
	}
	#pragma unroll
	for(int i=0;i<TSD;++i)
	{
		savedi[i]=distanceEUC_2D(ii[i].x,ii[i].y,ii[i+1].x,ii[i+1].y);
	}
	#pragma unroll
	for(int j=0;j<TSD;++j)
	{
		savedj[j]=distanceEUC_2D(jj[j].x,jj[j].y,jj[j+1].x,jj[j+1].y);
	}
	#pragma unroll
	for(int sx=0;sx<TSD+1;++sx)
	{
		#pragma unroll
		for(int sy=0;sy<TSD+1;++sy)
		{
			savedij[sx][sy]=distanceEUC_2D(ii[sx].x,ii[sx].y,jj[sy].x,jj[sy].y);
		}
	}
	//__syncthreads();

	//Now let us calculate the differences!
	dtype min = 0;
	int mink = -1;
	#pragma unroll
	for(int sx=0;sx<TSD;++sx)
	{
		#pragma unroll
		for(int sy=0;sy<TSD;++sy)
		{
			//first calculate the global indices to see if it is in calculation domain.
			int gi = istart+sx;
			int gj = jstart+sy;
			if(gi<maxn+1 && gj>gi && gj < nc-1-bj*BSD)
			{
				dtype dold = savedi[sx]+savedj[sy];
				dtype dnew = savedij[sx][sy]+savedij[sx+1][sy+1];
				//dtype dnew = distanceEUC_2D(ii[sx].x,ii[sx].y,jj[sy].x,jj[sy].y)+
				//		distanceEUC_2D(ii[sx+1].x,ii[sx+1].y,jj[sy+1].x,jj[sy+1].y);
				dtype difference = dnew-dold;
				mink = (difference<min)?(kfromij(gi,gj)):mink;
				min = (difference<min)?difference:min;
			}
		}
	}
	//__syncthreads();

	//Reduction step
	__shared__ dtype sdata[TPB2*TPB2];
	__shared__ int smink[TPB2*TPB2];
	int tid = threadIdx.x+threadIdx.y*TPB2;
	sdata[tid] = min;
	smink[tid] = mink;
	__syncthreads();
	int check=-1;
	dtype currentsdata = sdata[tid];
	dtype nextsdata;
	int currentsmink = smink[tid];
	int nextsmink;
	if (TPB2*TPB2 >= 1024){ if (tid < 512) {
		//check = sdata[tid+512]<sdata[tid];
		nextsdata = sdata[tid+512];
		nextsmink = smink[tid+512];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (TPB2*TPB2 >= 512) { if (tid < 256) {
		nextsdata = sdata[tid+256];
		nextsmink = smink[tid+256];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (TPB2*TPB2 >= 256) { if (tid < 128) {
		nextsdata = sdata[tid+128];
		nextsmink = smink[tid+128];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (TPB2*TPB2 >= 128) { if (tid < 64) {
		nextsdata = sdata[tid+64];
		nextsmink = smink[tid+64];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (tid < 32) warpMinReduce2(sdata,smink,tid);
	if (tid == 0){
		d_differences[block] = sdata[0];
		d_mink[block] = smink[0];
	}
}




__global__ void
gpuSwapReduce (dtype *d_differences,dtype *d_differences2,int *d_mink,int *d_mink2,int *d_ij,int nd)
{//Reduce to find minimum difference.
	//Then use the index to get the minimum k-index.
	__shared__ dtype sdata[REDUCE_TPB];
	__shared__ int smink[REDUCE_TPB];

	int tid = threadIdx.x;
	int gid = threadIdx.x+blockIdx.x*blockDim.x;
	if (gid < nd)
	{
		sdata[tid] = d_differences[gid];
		smink[tid] = d_mink[gid];
	}
	else
	{
		sdata[tid] = 0;
		smink[tid] = 0;
	}
	__syncthreads();
	int check=-1;
	dtype currentsdata = sdata[tid];
	dtype nextsdata;
	int currentsmink = smink[tid];
	int nextsmink;
	if (REDUCE_TPB >= 1024){ if (tid < 512) {
		//check = sdata[tid+512]<sdata[tid];
		nextsdata = sdata[tid+512];
		nextsmink = smink[tid+512];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (REDUCE_TPB >= 512) { if (tid < 256) {
		nextsdata = sdata[tid+256];
		nextsmink = smink[tid+256];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (REDUCE_TPB >= 256) { if (tid < 128) {
		nextsdata = sdata[tid+128];
		nextsmink = smink[tid+128];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (REDUCE_TPB >= 128) { if (tid < 64) {
		nextsdata = sdata[tid+64];
		nextsmink = smink[tid+64];
		check = nextsdata<currentsdata;
		sdata[tid] = currentsdata = (check)?nextsdata:currentsdata;
		smink[tid] = currentsmink = (check)?nextsmink:currentsmink; } 
		__syncthreads(); }
	if (tid < 32) warpMinReduce2(sdata,smink,tid);
	if (tid == 0){
		d_differences2[blockIdx.x] = sdata[0];
		d_mink2[blockIdx.x] = smink[0];
	}
	if((gridDim.x == 1) and (tid == 0))
	{
		d_ijfromk(smink[0],d_ij);
	}
}

__global__ void
gpuSwap (COORD *d_tour,int*d_ij)
{//The i,j indices are known beforehand.
	//and the blocks are configured knowing i,j (require host knowledge)
	int i = d_ij[0];
	int j = d_ij[1];
	int diff = j-i+1;
	int li = threadIdx.x + blockIdx.x*blockDim.x;
	int gi1 = i + li;
	int gi2 = j - li;
	if(li <= diff>>1)
	{
		COORD tmp1 = d_tour[gi1];
		COORD tmp2 = d_tour[gi2];
		d_tour[gi1] = tmp2;
		d_tour[gi2] = tmp1;
	}
}
__global__ void
gpuSwap2 (COORD *d_tour,int*d_ij)
{//The i,j indices are known beforehand.
	//blocks are NOT configured knowing i,j (which requires host knowledge)
	int gi = threadIdx.x + blockIdx.x*blockDim.x;
	int i = d_ij[0];
	int j2;
	if(gi >= i)
	{
		int j = d_ij[1];
		int diff = j-i+1;
		j2 = i+(diff>>1);
		if(gi <= j2)
		{
			int li = gi - i;
			int gi2 = j - li;
			COORD tmp1 = d_tour[gi];
			COORD tmp2 = d_tour[gi2];
			d_tour[gi] = tmp2;
			d_tour[gi2] = tmp1;
			/*
			if(gi==j2)
			{
				d_ij[0] = -42;
				d_ij[1] = -42;
			}
			*/
		}
	}
}

__global__ void
correctReduceBuffer (dtype *d_differences,dtype *d_differences2,int *d_mink,int *d_mink2)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	if(tid==0)
	{
		d_differences[0] = d_differences2[0];
		d_mink[0] = d_mink2[0];
	}
}


void
initCudaArray (dtype **d_A, dtype *h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (hipMalloc ((void**) d_A, N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (*d_A, h_A, N * sizeof (dtype),hipMemcpyHostToDevice));
}

void
initCudaTour (int **d_A, int *h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (hipMalloc ((void**) d_A, N * sizeof (int)));
	CUDA_CHECK_ERROR (hipMemcpy (*d_A, h_A, N * sizeof (int),hipMemcpyHostToDevice));
}

void transferCudaCoords(COORD*d_A, COORD*h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (
		hipMemcpy (d_A,h_A,N*sizeof(COORD),hipMemcpyHostToDevice)
		);
}
void initCudaCoords (COORD **d_A, COORD*h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (hipMalloc ((void**) d_A, N * sizeof (COORD)));
	transferCudaCoords(*d_A,h_A,N);
}
void getCudaCoords(COORD*d_A, COORD*h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (
		hipMemcpy (h_A,d_A,N*sizeof(COORD), hipMemcpyDeviceToHost)
		);
}
void getCudaInts(int*d_A, int*h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (
		hipMemcpy (h_A,d_A,N*sizeof(int), hipMemcpyDeviceToHost)
		);
}
void putCudaInts(int*d_A, int*h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (
		hipMemcpy (d_A,h_A,N*sizeof(int),hipMemcpyHostToDevice)
		);
}
void getCudaDtype(dtype*d_A, dtype*h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (
		hipMemcpy (h_A,d_A,N*sizeof(dtype), hipMemcpyDeviceToHost)
		);
}
void gpuSwapCheck(COORD* h_tour,int nc)
{//swaps h_tour.
	COORD* d_tour;
	initCudaCoords(&d_tour,h_tour,nc);
	
	int ij[2] = { nc/5, 5*nc/6 };
	int* d_ij;
	initCudaTour(&d_ij,ij,2);
	
	int tpb = 256;
	int blocks;
	//require host knowledge
	int diff = ij[1]-ij[0];
	blocks = (diff+tpb-1)/tpb;
	dim3 grid(blocks,1);
	dim3 block(tpb,1);
	//gpuSwap<<<grid,block>>>(d_tour,d_ij);
	//end
	//dont require host knowledge
	blocks = (nc+tpb-1)/tpb;
	dim3 grid2(blocks,1);
	dim3 block2(tpb,1);
	gpuSwap2<<<grid2,block2>>>(d_tour,d_ij);
	hipDeviceSynchronize();

	swapCoords(h_tour,ij);

	COORD tourFromGpu[nc];
	getCudaCoords(d_tour,tourFromGpu,nc);

	fprintf(stderr,"Comparing the gpu and cpu swap implementations...\n");
	for(int i=0;i<nc;++i)
	{
		if((tourFromGpu[i].x != h_tour[i].x) or (tourFromGpu[i].y != h_tour[i].y))
		{
			fprintf(stderr,"Mismatch detected at index %d!\n",i);
		}
	}
	fprintf(stderr,"Done!\n");

	hipFree(d_tour);
	hipFree(d_ij);
}

void checkCoordValidity(COORD*newTour,COORD*originalTour,int nc)
{
	fprintf(stderr,"Checking the validity of the new tour...\n");
	int occurences[nc];
	int repetitions[nc];
	//initialize occurences.
	for(int i=0;i<nc;++i)
	{
		occurences[i] = 0;
		repetitions[i] = -1;
	}
	//repetition within new tour.
	for(int i=0;i<nc;++i)
	{
		dtype x = newTour[i].x;
		dtype y = newTour[i].y;
		for(int j=0;j<nc;++j)
		{
			if((x == newTour[j].x) and (y == newTour[j].y))
			{
				++repetitions[i];
			}
		}
		if(repetitions[i] > 0)
		{
			fprintf(stderr,"Error! City %d is repeated in new tour!\n",i);
		}
	}
	//check that every city in original tour is accounted for.
	for(int i=0;i<nc;++i)
	{
		dtype x = newTour[i].x;
		dtype y = newTour[i].y;
		//go through the original tour.
		for(int j=0;j<nc;++j)
		{
			if((x == originalTour[j].x) and (y == originalTour[j].y))
			{
				++occurences[i];
			}
		}
		if(occurences[i] > 1)
		{
			fprintf(stderr,"Error! City %d was used more than once!\n",i);
		}
		if(occurences[i] < 1)
		{
			fprintf(stderr,"Error! City %d was not used!\n",i);
		}
	}
	fprintf(stderr,"Tour validity check done!\n");
}

typedef struct bestimprovement
{
	int k;
	dtype diff;
} BESTIMPROVEMENT;

BESTIMPROVEMENT minReduceCpu2(dtype*h,dtype*d,int*h_mink,int*d_mink,int n)
{
	CUDA_CHECK_ERROR (hipMemcpy (h, d, n * sizeof (dtype), hipMemcpyDeviceToHost));
	CUDA_CHECK_ERROR (hipMemcpy (h_mink, d_mink, n * sizeof (int), hipMemcpyDeviceToHost));
	//A minimum reduction.
	dtype minDifference=0;
	int minI = -1;
	for(int i=0; i<n; i++)
	{
		if(h[i] < minDifference)
		{
			minDifference = h[i];
			minI = h_mink[i];
		}
	}
	BESTIMPROVEMENT ret = {minI,minDifference};
	return ret;
}

void swapDeviceTour(COORD*h_tour,COORD*d_tour,int nc,int swapij[])
{//swaps host tour, then uploads to device.
	swapCoords(h_tour,swapij);
	transferCudaCoords(d_tour,h_tour,nc);
}
dtype coordDistance(COORD c1,COORD c2)
{
	dtype dx = c2.x-c1.x;
	dtype dy = c2.y-c1.y;
	return sqrt(dx*dx+dy*dy);
}
dtype tourLength(COORD*h_tour,int nc)
{
	dtype total=0;
	for(int i=0;i<nc;++i)
	{
		total+=coordDistance(h_tour[i],h_tour[(i+1)%nc]);
	}
	return total;
}
void cudaOpt2 (COORD *h_coords,COORD *d_coords, unsigned int nc)
{//d_coords and h_coords evolve.
	//Set and determine thread and block dimensions
	int N = ((nc-3)*(nc-2))/2;//Number of swaps to check (each thread does >= 1)
	//int NT = N;//Number of Threads.
	int ND = -1;//Number of differences output from GPU to min-reduce. 
	int GX=1,GY=1;//block dimensions of grid.
	int BX=1,BY=1;//thread dimensions of block.
	int NN = nc-3;//such that N = NN*(NN+1)/2
	int SBB = TPB2*TSD;//Block dimension  terms of swaps. SBB = BB if swaps per thread is 1.
	int BB2 =(NN+SBB-1)/SBB;///Block dimension of the grid, for TPB2-sized square blocks WITH TSD^2 swaps performed per thread (versus regular 1).
	int gridSplits = -1;//for when the problem gets too large for 65535 blocks.
	int lastGX = -1;//if gridSplits > 0, this is the number of blocks in the last split (less than BLOCKLIMIT)
	int totalBlocks = -1;
	
		GX = BB2*(BB2+1)/2;
		BX = TPB2;
		BY = TPB2;
		ND = GX;
		totalBlocks = BB2*(BB2+1)/2;
		//int swapChecksPerThread = TSD*TSD;
	
	//int swapChecksPerBlock = BX*BY*swapChecksPerThread;
	gridSplits = totalBlocks/BLOCKLIMIT;

	GX=BLOCKLIMIT;
	dim3 grid (GX,GY);
	dim3 block (BX,BY);
	lastGX = totalBlocks-gridSplits*BLOCKLIMIT;
	dim3 lastGrid (lastGX,GY);

	//Preprocessing for the gpu second reduction
	int cityblocks = (nc+SWAP_TPB-1)/SWAP_TPB;
	dim3 grid2 (cityblocks,1);
	dim3 block2 (SWAP_TPB,1);
	//end

	//Output information about run time parameters.
	fprintf (stderr, "Total Number of Blocks: %d\n",totalBlocks);
	fprintf (stderr, "Limited Grid Dim: %d x %d\n",GX,GY);
	fprintf (stderr, "Last Grid Dim: %d x %d\n",lastGX,GY);
	fprintf (stderr, "Block Dim: %d x %d\n",BX,BY);
	fprintf (stderr, "Size of Returned Array: %d\n",ND);
	fprintf (stderr, "Number of swaps computed and reduced: %d\n",N);
	//End output

	//Resource allocation.
	//dtype h_differences[ND];
	dtype *h_differences = new dtype[ND]; 
	dtype *d_differences,*d_differences2;
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_differences, ND * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_differences2, ND * sizeof (dtype)));
	int *h_mink = new int[ND]; 
	int *d_mink,*d_mink2;
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_mink, ND * sizeof (int)));
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_mink2, ND * sizeof (int)));
	int *d_ij;
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_ij, 2 * sizeof (int)));
	//hipDeviceSetCacheConfig(hipFuncCachePreferL1);//Shared Memory <-> L1 Cache setting.

	//Copy for correctness check.
	COORD original[nc];
	for(int i=0;i<nc;++i) { original[i] = h_coords[i]; }
	//Copy end.

	//gpu swap check
	//COORD h_swapcheck[nc];
	//for(int i=0;i<nc;++i) { h_swapcheck[i] = h_coords[i]; }
	//gpuSwapCheck(h_swapcheck,nc);
	//end check

	//Timer initialization and start
	hipEvent_t start, stop;
	CUDA_CHECK_ERROR (hipEventCreate (&start));
	CUDA_CHECK_ERROR (hipEventCreate (&stop));
	CUDA_CHECK_ERROR (hipEventRecord (start, 0));
	//end

	//EXECUTE KERNEL
	BESTIMPROVEMENT bi = {0,0};
	//Run the kernel for some iterations
	//dtype ptl = tourLength(h_coords,nc);
	int iterations;
	for(iterations = 0; iterations < MAX_ITER; ++iterations) {
		//Single best-improvement iteration
		hipDeviceSynchronize();
		for(int j=0;j<gridSplits;++j)
		{
			kernel11 <<<grid, block>>> (d_coords, d_differences,d_mink,
				nc,BB2,j*BLOCKLIMIT);
		}
		kernel11 <<<lastGrid, block>>> (d_coords, d_differences,d_mink,
			nc,BB2,gridSplits*BLOCKLIMIT);
		//end single iteration
		
		//Compute and display tour length.
		//dtype tl = tourLength(h_coords,nc);
		//fprintf (stderr, "Current tour length %f (difference from previous: %f)\n",tl,tl-ptl);
		//ptl = tl;
		//end

/*
		//cpu reduce
		hipDeviceSynchronize();
		bi = minReduceCpu2(h_differences,d_differences,h_mink,d_mink,ND);
		int ij[2];
		ijfromk(bi.k,ij);
		putCudaInts(d_ij,ij,2);
		//end
*/

		//gpu reduce
		int elements = ND;
		int mode = 1;
		while(elements > 1)
		{
			//fprintf(stderr,"elements: %d\n",elements);
			int rblocks = (elements+REDUCE_TPB-1)/REDUCE_TPB;
			dim3 rgrid (rblocks,1);
			dim3 rblock (REDUCE_TPB,1);
			hipDeviceSynchronize();
			//fprintf(stderr,"rblocks: %d\n",rblocks);
			if(mode==1)
			{
				gpuSwapReduce<<<rgrid,rblock>>>(d_differences,d_differences2,d_mink,d_mink2,d_ij,elements);
				mode=0;
			}
			else
			{
				gpuSwapReduce<<<rgrid,rblock>>>(d_differences2,d_differences,d_mink2,d_mink,d_ij,elements);
				mode=1;
			}
			elements=rblocks;
		}
		/*
		if(mode==0)
		{
			dim3 grid3 (1,1);
			dim3 block3 (1,1);
			hipDeviceSynchronize();
			correctReduceBuffer<<<grid3,block3>>>(d_differences,d_differences2,d_mink,d_mink2);
		}
		*/
		//end

		//gpu tour rearrangement
		hipDeviceSynchronize();
		//fprintf(stderr,"grid2, block2: %d, %d\n",cityblocks,SWAP_TPB);
		gpuSwap2<<<grid2,block2>>>(d_coords,d_ij);
		//end

		//post-processing and optional cpu tour rearrangement
		if(iterations%REFRESH_ITER==0)
		{
			hipDeviceSynchronize();
			getCudaDtype(d_differences,&bi.diff,1);
			getCudaInts(d_mink,&bi.k,1);
			int ij[2];
			getCudaInts(d_ij,ij,2);
			hipDeviceSynchronize();
			//int ij[2];
			//ijfromk(bi.k,ij);
			if(bi.diff < 0)
			{
				//cpu tour rearrangement
				//swapDeviceTour(h_coords,d_coords,nc,ij);
				fprintf (stderr, "Iteration %d GPU min diff,i,j,raw k : %f,%d,%d,%d\n",iterations,bi.diff,ij[0],ij[1],bi.k);
			}
			else
			{
				fprintf (stderr, "No more improvements found! Stopping iteration.\n"); 
				break;
			}
		}
		//end
	}
	//END KERNEL EXECUTION

	//Timer stop, get time, destroy timer resources
	CUDA_CHECK_ERROR (hipEventRecord (stop, 0));
	CUDA_CHECK_ERROR (hipEventSynchronize (stop));
	float elapsedTimePerIteration;
	CUDA_CHECK_ERROR (hipEventElapsedTime (&elapsedTimePerIteration, start, stop));
	elapsedTimePerIteration /= iterations;
	CUDA_CHECK_ERROR (hipEventDestroy (start));
	CUDA_CHECK_ERROR (hipEventDestroy (stop));

	//Check tour correctness.
	checkCoordValidity(h_coords,original,nc);
	//end

	//Output the timing and performance results.
	fprintf (stderr, "GPU min diff, raw k : %f, %d\n",bi.diff,bi.k);
	fprintf (stderr, "Execution time per iteration: %f ms\n", elapsedTimePerIteration);
	fprintf (stderr, "Speed (Gmoves/s): %f\n\n",N/(elapsedTimePerIteration/1000.0)/1e9);
	fprintf (stderr, "Threads Per Block: %d x %d\n",BX,BY);
	//fprintf (stderr, "Equivalent performance: %f GB/s\n", (N * sizeof (dtype) / elapsedTime) * 1e-6);
}